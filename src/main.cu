#include <iostream>
#include "precompute.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

int main() {
    int32_t depth = 16;
    int32_t primes[30];
    dgt_context* dgtContext[30];
    for(int i=0; i < 30; i++){
        primes[i] = 257;
    }
    for(int i=0; i < 30; i++){
        if(!generate_dgt_context(primes[i],depth,&dgtContext[i])){
            std::cout<<"fail to generate dgt context for the"<<i<<"-th prime"<<std::endl;
        }
    }




    return 0;
}